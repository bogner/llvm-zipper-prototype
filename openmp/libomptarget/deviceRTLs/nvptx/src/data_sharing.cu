#include "hip/hip_runtime.h"
//===----- data_sharing.cu - NVPTX OpenMP debug utilities -------- CUDA -*-===//
//
//                     The LLVM Compiler Infrastructure
//
// This file is dual licensed under the MIT and the University of Illinois Open
// Source Licenses. See LICENSE.txt for details.
//
//===----------------------------------------------------------------------===//
//
// This file contains the implementation of data sharing environments/
//
//===----------------------------------------------------------------------===//
#include "omptarget-nvptx.h"
#include <stdio.h>

// Number of threads in the CUDA block.
__device__ static unsigned getNumThreads() { return blockDim.x; }
// Thread ID in the CUDA block
__device__ static unsigned getThreadId() { return threadIdx.x; }
// Warp ID in the CUDA block
__device__ static unsigned getWarpId() { return threadIdx.x / WARPSIZE; }

// The CUDA thread ID of the master thread.
__device__ static unsigned getMasterThreadId() {
  unsigned Mask = WARPSIZE - 1;
  return (getNumThreads() - 1) & (~Mask);
}

// Find the active threads in the warp - return a mask whose n-th bit is set if
// the n-th thread in the warp is active.
__device__ static unsigned getActiveThreadsMask() {
  return __BALLOT_SYNC(0xFFFFFFFF, true);
}

// Return true if this is the first active thread in the warp.
__device__ static bool IsWarpMasterActiveThread() {
  unsigned long long Mask = getActiveThreadsMask();
  unsigned long long ShNum = WARPSIZE - (getThreadId() % WARPSIZE);
  unsigned long long Sh = Mask << ShNum;
  // Truncate Sh to the 32 lower bits
  return (unsigned)Sh == 0;
}
// Return true if this is the master thread.
__device__ static bool IsMasterThread() {
  return getMasterThreadId() == getThreadId();
}

/// Return the provided size aligned to the size of a pointer.
__device__ static size_t AlignVal(size_t Val) {
  const size_t Align = (size_t)sizeof(void *);
  if (Val & (Align - 1)) {
    Val += Align;
    Val &= ~(Align - 1);
  }
  return Val;
}

#define DSFLAG 0
#define DSFLAG_INIT 0
#define DSPRINT(_flag, _str, _args...)                                         \
  {                                                                            \
    if (_flag) {                                                               \
      /*printf("(%d,%d) -> " _str, blockIdx.x, threadIdx.x, _args);*/          \
    }                                                                          \
  }
#define DSPRINT0(_flag, _str)                                                  \
  {                                                                            \
    if (_flag) {                                                               \
      /*printf("(%d,%d) -> " _str, blockIdx.x, threadIdx.x);*/                 \
    }                                                                          \
  }

// Initialize the shared data structures. This is expected to be called for the
// master thread and warp masters. \param RootS: A pointer to the root of the
// data sharing stack. \param InitialDataSize: The initial size of the data in
// the slot.
EXTERN void
__kmpc_initialize_data_sharing_environment(__kmpc_data_sharing_slot *rootS,
                                           size_t InitialDataSize) {

  DSPRINT0(DSFLAG_INIT,
           "Entering __kmpc_initialize_data_sharing_environment\n");

  unsigned WID = getWarpId();
  DSPRINT(DSFLAG_INIT, "Warp ID: %d\n", WID);

  omptarget_nvptx_TeamDescr *teamDescr =
      &omptarget_nvptx_threadPrivateContext->TeamContext();
  __kmpc_data_sharing_slot *RootS = teamDescr->RootS(WID);

  DataSharingState.SlotPtr[WID] = RootS;
  DataSharingState.StackPtr[WID] = (void *)&RootS->Data[0];

  // We don't need to initialize the frame and active threads.

  DSPRINT(DSFLAG_INIT, "Initial data size: %08x \n", InitialDataSize);
  DSPRINT(DSFLAG_INIT, "Root slot at: %016llx \n", (long long)RootS);
  DSPRINT(DSFLAG_INIT, "Root slot data-end at: %016llx \n",
          (long long)RootS->DataEnd);
  DSPRINT(DSFLAG_INIT, "Root slot next at: %016llx \n", (long long)RootS->Next);
  DSPRINT(DSFLAG_INIT, "Shared slot ptr at: %016llx \n",
          (long long)DataSharingState.SlotPtr[WID]);
  DSPRINT(DSFLAG_INIT, "Shared stack ptr at: %016llx \n",
          (long long)DataSharingState.StackPtr[WID]);

  DSPRINT0(DSFLAG_INIT, "Exiting __kmpc_initialize_data_sharing_environment\n");
}

EXTERN void *__kmpc_data_sharing_environment_begin(
    __kmpc_data_sharing_slot **SavedSharedSlot, void **SavedSharedStack,
    void **SavedSharedFrame, int32_t *SavedActiveThreads,
    size_t SharingDataSize, size_t SharingDefaultDataSize,
    int16_t IsOMPRuntimeInitialized) {

  DSPRINT0(DSFLAG, "Entering __kmpc_data_sharing_environment_begin\n");

  // If the runtime has been elided, used __shared__ memory for master-worker
  // data sharing.
  if (!IsOMPRuntimeInitialized)
    return (void *)&DataSharingState;

  DSPRINT(DSFLAG, "Data Size %016llx\n", SharingDataSize);
  DSPRINT(DSFLAG, "Default Data Size %016llx\n", SharingDefaultDataSize);

  unsigned WID = getWarpId();
  unsigned CurActiveThreads = getActiveThreadsMask();

  __kmpc_data_sharing_slot *&SlotP = DataSharingState.SlotPtr[WID];
  void *&StackP = DataSharingState.StackPtr[WID];
  void *&FrameP = DataSharingState.FramePtr[WID];
  int32_t &ActiveT = DataSharingState.ActiveThreads[WID];

  DSPRINT0(DSFLAG, "Save current slot/stack values.\n");
  // Save the current values.
  *SavedSharedSlot = SlotP;
  *SavedSharedStack = StackP;
  *SavedSharedFrame = FrameP;
  *SavedActiveThreads = ActiveT;

  DSPRINT(DSFLAG, "Warp ID: %d\n", WID);
  DSPRINT(DSFLAG, "Saved slot ptr at: %016llx \n", (long long)SlotP);
  DSPRINT(DSFLAG, "Saved stack ptr at: %016llx \n", (long long)StackP);
  DSPRINT(DSFLAG, "Saved frame ptr at: %016llx \n", (long long)FrameP);
  DSPRINT(DSFLAG, "Active threads: %08x \n", ActiveT);

  // Only the warp active master needs to grow the stack.
  if (IsWarpMasterActiveThread()) {
    // Save the current active threads.
    ActiveT = CurActiveThreads;

    // Make sure we use aligned sizes to avoid rematerialization of data.
    SharingDataSize = AlignVal(SharingDataSize);
    // FIXME: The default data size can be assumed to be aligned?
    SharingDefaultDataSize = AlignVal(SharingDefaultDataSize);

    // Check if we have room for the data in the current slot.
    const uintptr_t CurrentStartAddress = (uintptr_t)StackP;
    const uintptr_t CurrentEndAddress = (uintptr_t)SlotP->DataEnd;
    const uintptr_t RequiredEndAddress =
        CurrentStartAddress + (uintptr_t)SharingDataSize;

    DSPRINT(DSFLAG, "Data Size %016llx\n", SharingDataSize);
    DSPRINT(DSFLAG, "Default Data Size %016llx\n", SharingDefaultDataSize);
    DSPRINT(DSFLAG, "Current Start Address %016llx\n", CurrentStartAddress);
    DSPRINT(DSFLAG, "Current End Address %016llx\n", CurrentEndAddress);
    DSPRINT(DSFLAG, "Required End Address %016llx\n", RequiredEndAddress);
    DSPRINT(DSFLAG, "Active Threads %08x\n", ActiveT);

    // If we require a new slot, allocate it and initialize it (or attempt to
    // reuse one). Also, set the shared stack and slot pointers to the new
    // place. If we do not need to grow the stack, just adapt the stack and
    // frame pointers.
    if (CurrentEndAddress < RequiredEndAddress) {
      size_t NewSize = (SharingDataSize > SharingDefaultDataSize)
                           ? SharingDataSize
                           : SharingDefaultDataSize;
      __kmpc_data_sharing_slot *NewSlot = 0;

      // Attempt to reuse an existing slot.
      if (__kmpc_data_sharing_slot *ExistingSlot = SlotP->Next) {
        uintptr_t ExistingSlotSize = (uintptr_t)ExistingSlot->DataEnd -
                                     (uintptr_t)(&ExistingSlot->Data[0]);
        if (ExistingSlotSize >= NewSize) {
          DSPRINT(DSFLAG, "Reusing stack slot %016llx\n",
                  (long long)ExistingSlot);
          NewSlot = ExistingSlot;
        } else {
          DSPRINT(DSFLAG, "Cleaning up -failed reuse - %016llx\n",
                  (long long)SlotP->Next);
          free(ExistingSlot);
        }
      }

      if (!NewSlot) {
        NewSlot = (__kmpc_data_sharing_slot *)malloc(
            sizeof(__kmpc_data_sharing_slot) + NewSize);
        DSPRINT(DSFLAG, "New slot allocated %016llx (data size=%016llx)\n",
                (long long)NewSlot, NewSize);
      }

      NewSlot->Next = 0;
      NewSlot->DataEnd = &NewSlot->Data[NewSize];

      SlotP->Next = NewSlot;
      SlotP = NewSlot;
      StackP = &NewSlot->Data[SharingDataSize];
      FrameP = &NewSlot->Data[0];
    } else {

      // Clean up any old slot that we may still have. The slot producers, do
      // not eliminate them because that may be used to return data.
      if (SlotP->Next) {
        DSPRINT(DSFLAG, "Cleaning up - old not required - %016llx\n",
                (long long)SlotP->Next);
        free(SlotP->Next);
        SlotP->Next = 0;
      }

      FrameP = StackP;
      StackP = (void *)RequiredEndAddress;
    }
  }

  // FIXME: Need to see the impact of doing it here.
  __threadfence_block();

  DSPRINT0(DSFLAG, "Exiting __kmpc_data_sharing_environment_begin\n");

  // All the threads in this warp get the frame they should work with.
  return FrameP;
}

EXTERN void __kmpc_data_sharing_environment_end(
    __kmpc_data_sharing_slot **SavedSharedSlot, void **SavedSharedStack,
    void **SavedSharedFrame, int32_t *SavedActiveThreads,
    int32_t IsEntryPoint) {

  DSPRINT0(DSFLAG, "Entering __kmpc_data_sharing_environment_end\n");

  unsigned WID = getWarpId();

  if (IsEntryPoint) {
    if (IsWarpMasterActiveThread()) {
      DSPRINT0(DSFLAG, "Doing clean up\n");

      // The master thread cleans the saved slot, because this is an environment
      // only for the master.
      __kmpc_data_sharing_slot *S =
          IsMasterThread() ? *SavedSharedSlot : DataSharingState.SlotPtr[WID];

      if (S->Next) {
        free(S->Next);
        S->Next = 0;
      }
    }

    DSPRINT0(DSFLAG, "Exiting Exiting __kmpc_data_sharing_environment_end\n");
    return;
  }

  int32_t CurActive = getActiveThreadsMask();

  // Only the warp master can restore the stack and frame information, and only
  // if there are no other threads left behind in this environment (i.e. the
  // warp diverged and returns in different places). This only works if we
  // assume that threads will converge right after the call site that started
  // the environment.
  if (IsWarpMasterActiveThread()) {
    int32_t &ActiveT = DataSharingState.ActiveThreads[WID];

    DSPRINT0(DSFLAG, "Before restoring the stack\n");
    // Zero the bits in the mask. If it is still different from zero, then we
    // have other threads that will return after the current ones.
    ActiveT &= ~CurActive;

    DSPRINT(DSFLAG, "Active threads: %08x; New mask: %08x\n", CurActive,
            ActiveT);

    if (!ActiveT) {
      // No other active threads? Great, lets restore the stack.

      __kmpc_data_sharing_slot *&SlotP = DataSharingState.SlotPtr[WID];
      void *&StackP = DataSharingState.StackPtr[WID];
      void *&FrameP = DataSharingState.FramePtr[WID];

      SlotP = *SavedSharedSlot;
      StackP = *SavedSharedStack;
      FrameP = *SavedSharedFrame;
      ActiveT = *SavedActiveThreads;

      DSPRINT(DSFLAG, "Restored slot ptr at: %016llx \n", (long long)SlotP);
      DSPRINT(DSFLAG, "Restored stack ptr at: %016llx \n", (long long)StackP);
      DSPRINT(DSFLAG, "Restored frame ptr at: %016llx \n", (long long)FrameP);
      DSPRINT(DSFLAG, "Active threads: %08x \n", ActiveT);
    }
  }

  // FIXME: Need to see the impact of doing it here.
  __threadfence_block();

  DSPRINT0(DSFLAG, "Exiting __kmpc_data_sharing_environment_end\n");
  return;
}

EXTERN void *
__kmpc_get_data_sharing_environment_frame(int32_t SourceThreadID,
                                          int16_t IsOMPRuntimeInitialized) {
  DSPRINT0(DSFLAG, "Entering __kmpc_get_data_sharing_environment_frame\n");

  // If the runtime has been elided, use __shared__ memory for master-worker
  // data sharing.  We're reusing the statically allocated data structure
  // that is used for standard data sharing.
  if (!IsOMPRuntimeInitialized)
    return (void *)&DataSharingState;

  // Get the frame used by the requested thread.

  unsigned SourceWID = SourceThreadID / WARPSIZE;

  DSPRINT(DSFLAG, "Source  warp: %d\n", SourceWID);

  void *P = DataSharingState.FramePtr[SourceWID];
  DSPRINT0(DSFLAG, "Exiting __kmpc_get_data_sharing_environment_frame\n");
  return P;
}

////////////////////////////////////////////////////////////////////////////////
// Runtime functions for trunk data sharing scheme.
////////////////////////////////////////////////////////////////////////////////

// Initialize data sharing data structure. This function needs to be called
// once at the beginning of a data sharing context (coincides with the kernel
// initialization).
EXTERN void __kmpc_data_sharing_init_stack() {
  // This function initializes the stack pointer with the pointer to the
  // statically allocated shared memory slots. The size of a shared memory
  // slot is pre-determined to be 256 bytes.
  unsigned WID = getWarpId();
  omptarget_nvptx_TeamDescr *teamDescr =
      &omptarget_nvptx_threadPrivateContext->TeamContext();
  __kmpc_data_sharing_slot *RootS = teamDescr->RootS(WID);

  DataSharingState.SlotPtr[WID] = RootS;
  DataSharingState.StackPtr[WID] = (void *)&RootS->Data[0];

  // We initialize the list of references to arguments here.
  omptarget_nvptx_globalArgs.Init();
}

// Called at the time of the kernel initialization. This is used to initilize
// the list of references to shared variables and to pre-allocate global storage
// for holding the globalized variables.
//
// By default the globalized variables are stored in global memory. If the
// UseSharedMemory is set to true, the runtime will attempt to use shared memory
// as long as the size requested fits the pre-allocated size.
//
// TODO: allow more than one push per slot to save on calls to malloc.
// Currently there is only one slot for each push so the data size in the slot
// is the same size as the size being requested.
//
// Called by: master, TODO: call by workers
EXTERN void* __kmpc_data_sharing_push_stack(size_t size,
    int16_t UseSharedMemory) {
  // TODO: Add shared memory support. For now, use global memory only for
  // storing the data sharing slots so ignore the pre-allocated
  // shared memory slot.

  // Use global memory for storing the stack.
  if (IsMasterThread()) {
    unsigned WID = getWarpId();

    // SlotP will point to either the shared memory slot or an existing
    // global memory slot.
    __kmpc_data_sharing_slot *&SlotP = DataSharingState.SlotPtr[WID];
    __kmpc_data_sharing_slot *&TailSlotP = DataSharingState.TailPtr[WID];

    // The slot for holding the data we are pushing.
    __kmpc_data_sharing_slot *NewSlot = 0;
    size_t NewSize = size;

    // Check if there is a next slot.
    if (__kmpc_data_sharing_slot *ExistingSlot = SlotP->Next) {
      // Attempt to re-use an existing slot provided the data fits in the slot.
      // The leftover data space will not be used.
      ptrdiff_t ExistingSlotSize = (uintptr_t)ExistingSlot->DataEnd -
                                   (uintptr_t)(&ExistingSlot->Data[0]);
      if (ExistingSlotSize >= NewSize)
        NewSlot = ExistingSlot;
      else
        free(ExistingSlot);
    }

    if (!NewSlot) {
      NewSlot = (__kmpc_data_sharing_slot *)malloc(
          sizeof(__kmpc_data_sharing_slot) + NewSize);
      NewSlot->Next = 0;
      NewSlot->Prev = SlotP;

      // This is the last slot, save it.
      TailSlotP = NewSlot;
    }

    NewSlot->DataEnd = &NewSlot->Data[NewSize];

    SlotP->Next = NewSlot;
    SlotP = NewSlot;

    return (void*)&SlotP->Data[0];
  }

  // TODO: add memory fence here when this function can be called by
  // worker threads also. For now, this function is only called by the
  // master thread of each team.

  // TODO: implement sharing across workers.
  return 0;
}

// Pop the stack and free any memory which can be reclaimed.
//
// When the pop operation removes the last global memory slot,
// reclaim all outstanding global memory slots since it is
// likely we have reached the end of the kernel.
EXTERN void __kmpc_data_sharing_pop_stack(void *a) {
  if (IsMasterThread()) {
    unsigned WID = getWarpId();

    __kmpc_data_sharing_slot *S = DataSharingState.SlotPtr[WID];

    if (S->Prev)
      S = S->Prev;

    // If this will "pop" the last global memory node then it is likely
    // that we are at the end of the data sharing region and we can
    // de-allocate any existing global memory slots.
    if (!S->Prev) {
      __kmpc_data_sharing_slot *Tail = DataSharingState.TailPtr[WID];

      while(Tail && Tail->Prev) {
        Tail = Tail->Prev;
        free(Tail->Next);
        Tail->Next=0;
      }
    }

    return;
  }

  // TODO: add memory fence here when this function can be called by
  // worker threads also. For now, this function is only called by the
  // master thread of each team.

  // TODO: implement sharing across workers.
}

// Begin a data sharing context. Maintain a list of references to shared
// variables. This list of references to shared variables will be passed
// to one or more threads.
// In L0 data sharing this is called by master thread.
// In L1 data sharing this is called by active warp master thread.
EXTERN void __kmpc_begin_sharing_variables(void ***GlobalArgs, size_t nArgs) {
  omptarget_nvptx_globalArgs.EnsureSize(nArgs);
  *GlobalArgs = omptarget_nvptx_globalArgs.GetArgs();
}

// End a data sharing context. There is no need to have a list of refs
// to shared variables because the context in which those variables were
// shared has now ended. This should clean-up the list of references only
// without affecting the actual global storage of the variables.
// In L0 data sharing this is called by master thread.
// In L1 data sharing this is called by active warp master thread.
EXTERN void __kmpc_end_sharing_variables() {
  omptarget_nvptx_globalArgs.DeInit();
}

// This function will return a list of references to global variables. This
// is how the workers will get a reference to the globalized variable. The
// members of this list will be passed to the outlined parallel function
// preserving the order.
// Called by all workers.
EXTERN void __kmpc_get_shared_variables(void ***GlobalArgs) {
  *GlobalArgs = omptarget_nvptx_globalArgs.GetArgs();
}
